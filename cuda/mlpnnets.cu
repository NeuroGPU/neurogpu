#include "hip/hip_runtime.h"
/*

  mlpnnets.cu
  Implementation of feedforward MLP neural networks in CUDA.

  Andrei de A. Formiga, 2012-05-09

 */

#include <stdio.h>
#include <stdlib.h>

#include "mlpnnets.h"


// --- utility functions --------------------------------------------------
inline float* allocateFloatsDev(int n)
{
    float *res;

    if (hipMalloc((void**) &res, n * sizeof(float)) != hipSuccess) {
        return NULL;
    }

    return res;
}

// --- activation functions -----------------------------------------------

// sigmoid activation function
__device__ float asigmoid(float t)
{
    return 1.0f / (1.0f + expf(-t));
}

__device__ float dsigmoid(float output)
{
    return output * (1.0f - output);
}


// --- initialization -----------------------------------------------------

// make randomly generated weights in (0.0, 1.0] be in the
// interval from -max_abs to +max_abs
__global__ void normalize_weights(float *w, float max_abs)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    w[tid] = ((w[tid] - 0.5f) / 0.5f) * max_abs;
}

// random initialization for weights
// w must be an array of floats on the device
void RandomWeights(MLPNetwork *net, float max_abs, long seed)
{
    hiprandGenerator_t gen;

    // create and initialize generator
    hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_XORWOW);
    hiprandSetPseudoRandomGeneratorSeed(gen, seed);
    hiprandSetGeneratorOrdering(gen, HIPRAND_ORDERING_PSEUDO_SEEDED);

    hiprandGenerateUniform(gen, net->d_weights, net->nWeights);
    normalize_weights<<<1, net->nWeights>>>(net->d_weights, max_abs);
    hiprandDestroyGenerator(gen);
}

// initialize weights randomly using the supplied generator
// w must be an array of floats on the device
void RandomWeightsGen(MLPNetwork *net, float max_abs, hiprandGenerator_t gen)
{
    hiprandGenerateUniform(gen, net->d_weights, net->nWeights);
    normalize_weights<<<1, net->nWeights>>>(net->d_weights, max_abs);
}


// --- network construction and management --------------------------------
void DestroyLayer(MLPLayer *layer)
{
    if (layer->d_outs != NULL) {
        hipFree(layer->d_outs);
        layer->d_outs = NULL;
    }

    if (layer->d_deltas != NULL) {
        hipFree(layer->d_deltas);
        layer->d_deltas = NULL;
    }

    free(layer);
}

// free all memory on device reserved for deltas, on all layers
void FreeDeltas(MLPNetwork *nnet)
{
    for (int i = 1; i < nnet->nLayers; ++i) {
        if (nnet->layers[i]->d_deltas != NULL) {
            hipFree(nnet->layers[i]->d_deltas);
            nnet->layers[i]->d_deltas = NULL;
        }
    }
}

void FreeOutputs(MLPNetwork *nnet)
{
    // do not free memory for layer 0 outputs (they come from inputs)
    for (int i = 1; i < nnet->nLayers; ++i) {
        if (nnet->layers[i]->d_outs != NULL) {
            hipFree(nnet->layers[i]->d_outs);
            nnet->layers[i]->d_outs = NULL;
        }
    }    
}

// allocates memory on device for outputs on all layers
// assumes the number of cases is already net on the nnet object
bool ReallocateOutputs(MLPNetwork *nnet)
{
    // free outputs if already allocated
    FreeOutputs(nnet);

    // allocate memory for outputs (don't allocate for layer 0)
    for (int i = 1; i < nnet->nLayers; ++i) {
        nnet->layers[i]->d_outs =
            allocateFloatsDev(nnet->layers[i]->nNeurons * nnet->nCases);
        if (nnet->layers[i]->d_outs == NULL) {
            FreeOutputs(nnet);
            return false;
        }                
    }

    return true;
}

// allocates memory on device for the deltas on all layers
// assumes the number of cases is already set on the nnet object
bool ReallocateDeltas(MLPNetwork *nnet)
{
    // free deltas if already allocated
    FreeDeltas(nnet);

    // allocate memory for deltas, except for input layer
    for (int i = 1; i < nnet->nLayers; ++i) {
        nnet->layers[i]->d_deltas =
            allocateFloatsDev(nnet->layers[i]->nNeurons * nnet->nCases);
        if (nnet->layers[i]->d_deltas == NULL) {
            FreeDeltas(nnet);
            return false;
        }
    }

    return true;
}

MLPLayer *CreateLayer(int nNeurons, int nNeuronsPrev, int wOffset)
{
    MLPLayer *result = (MLPLayer*) calloc(1, sizeof(MLPLayer));

    if (result == NULL)
        return NULL;

    result->nNeurons = nNeurons;

    // mark outputs and deltas as not allocated
    result->d_outs = NULL;
    result->d_deltas = NULL;

    result->weightsPerNeuron = nNeuronsPrev + 1;
    result->weightOffset = wOffset;

    return result;
}

// Create a MLP neural network for execution on the GPU.
// nLayers: number of layers
// neuronsPerLayer: array of ints (size equal to nLayers) with the
//                  number of neurons for each layer
MLPNetwork *CreateNetwork(int nLayers, int *neuronsPerLayer)
{
    MLPNetwork *result;

    result = (MLPNetwork*) calloc(1, sizeof(MLPNetwork));
    
    if (result == NULL)
        return NULL;

    // network is not initially prepared to store outputs, so zero input cases
    result->nCases = 0;
    
    result->nLayers = nLayers;
    result->layers = (MLPLayer**) calloc(nLayers, sizeof(MLPLayer*));

    if (result->layers == NULL) {
        free(result);
        return NULL;
    }

    // create input layer
    result->layers[0] = CreateLayer(neuronsPerLayer[0], 0, 0);
    if (result->layers[0] == NULL) {
        DestroyNetwork(result);
        return NULL;
    }

    // create remaining layers, and sum the number of weights
    int nwTotal = 0;
    int nwPrev = neuronsPerLayer[0];        
    for (int i = 1; i < nLayers; ++i) {
        result->layers[i] = CreateLayer(neuronsPerLayer[i], nwPrev, nwTotal);
        if (result->layers[i] == NULL) {
            DestroyNetwork(result);
            return NULL;
        }

        nwTotal += neuronsPerLayer[i] * (nwPrev + 1);
        nwPrev = neuronsPerLayer[i];        
    }

    result->nWeights = nwTotal;
    result->d_weights = allocateFloatsDev(result->nWeights);

    if (result->d_weights == NULL) {
        DestroyNetwork(result);
        return NULL;
    }        

    return result;
}

void DestroyNetwork(MLPNetwork *net)
{
    if (net->d_weights != NULL) {
        hipFree(net->d_weights);
        net->d_weights = NULL;
    }

    if (net->layers != NULL) {
        for (int i = 0; i < net->nLayers; ++i)
            if (net->layers[i] != NULL)
                DestroyLayer(net->layers[i]);

        free(net->layers);
        net->layers = NULL;
    }

    free(net);
}

DataSet* CreateDataSet(int nCases, int inputSize, int outputSize)
{
    DataSet *result;

    result = (DataSet*) malloc(sizeof(DataSet));

    if (result == NULL)
        return NULL;

    result->nCases = nCases;
    result->inputSize = inputSize;
    result->outputSize = outputSize;

    result->inputs = (float*) malloc(sizeof(float) * nCases * inputSize);

    if (result->inputs == NULL) {
        free(result);
        return NULL;
    }

    result->outputs = (float*) malloc(sizeof(float) * nCases * outputSize);

    if (result->outputs == NULL) {
        free(result->inputs);
        free(result);
        return NULL;
    }

    result->location = LOC_HOST;
    result->d_inputs = NULL;
    result->d_outputs = NULL;

    return result;
}

void DestroyDataSet(DataSet *dset)
{
    if (dset->inputs != NULL) {
        free(dset->inputs);
        dset->inputs = NULL;
    }

    if (dset->outputs != NULL) {
        free(dset->outputs);
        dset->outputs = NULL;
    }

    if (dset->location == LOC_HOST) {
        if (dset->d_inputs != NULL || dset->d_outputs != NULL) {
            fprintf(stderr, "Location of dataset is HOST but device ptrs are not NULL\n");
            exit(-1);
        }
    }
    else {
        if (dset->d_inputs != NULL) {
            hipFree(dset->d_inputs);
            dset->d_inputs = NULL;
        }

        if (dset->d_outputs != NULL) {
            hipFree(dset->d_outputs);
            dset->d_outputs = NULL;
        }
    }

    free(dset);
}

bool TransferDataSetToDevice(DataSet *data)
{
    hipError_t e;
    
    if (data->location == LOC_HOST) {
        int nFloatsIn = data->nCases * data->inputSize;
        int nFloatsOut = data->nCases * data->outputSize;

        // allocate memory for dataset in device
        data->d_inputs = allocateFloatsDev(nFloatsIn);

        if (data->d_inputs == NULL) 
            return false;

        data->d_outputs = allocateFloatsDev(nFloatsOut);

        if (data->d_outputs == NULL) {
            hipFree(data->d_inputs);
            data->d_inputs = NULL;
            return false;
        }

        // copy dataset to device
        e = hipMemcpy(data->d_inputs, data->inputs,
                       nFloatsIn * sizeof(float), hipMemcpyHostToDevice);

        if (e != hipSuccess) {
            fprintf(stderr, "Error copying dataset inputs from host to device: %s\n",
                    hipGetErrorString(e));
            return false;
        }
        
        e = hipMemcpy(data->d_outputs, data->outputs,
                       nFloatsOut * sizeof(float), hipMemcpyHostToDevice);

        if (e != hipSuccess) {
            fprintf(stderr, "Error copying dataset outputs from host to device: %s\n",
                    hipGetErrorString(e));
            return false;
        }
        
        // change location specifier
        data->location = LOC_BOTH;
    }

    return true;
}


// ------------------------------------------------------------------------
// --- forward propagation ------------------------------------------------
// ------------------------------------------------------------------------

// calculate outputs of one layer, assuming the previous
// layer was already calculated; the outputs corresponding to
// all input cases are computed in parallel
//
// grid will be <<<Nc, Nn>>> for Nc input cases and Nn neurons in layer
__global__ void forward_layer(float *d_weights, int weightOffset, int weightsPerNeuron,
                              float *d_ins, int neuronsPrev, float *d_outs)
{
    // weightsPerNeuron is always = to neuronsPrev+1
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int ixIn = blockIdx.x * neuronsPrev;
    int wid = weightOffset + (threadIdx.x * weightsPerNeuron);

    // bias input
    float a = d_weights[wid];

    for (int i = 1; i < weightsPerNeuron; ++i)
        a += d_weights[wid + i] * d_ins[ixIn + i-1];

    d_outs[tid] = asigmoid(a);
}

// calculate outputs of one layer using a threshold activation,
// assuming the previous layer was already calculated; the outputs
// corresponding to all input cases are computed in parallel
//
// grid will be <<<Nc, Nn>>> for Nc input cases and Nn neurons in layer
__global__ void forward_layer_threshold(float *d_weights, int weightOffset,
                                        int weightsPerNeuron,
                                        float *d_ins, int neuronsPrev,
                                        float *d_outs)
{
    // weightsPerNeuron is always = to neuronsPrev+1
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int ixIn = blockIdx.x * neuronsPrev;
    int wid = weightOffset + (threadIdx.x * weightsPerNeuron);

    // bias input
    float a = d_weights[wid];

    for (int i = 1; i < weightsPerNeuron; ++i)
        a += d_weights[wid + i] * d_ins[ixIn + i-1];

    d_outs[tid] = (a > 0.0f? 1.0f : 0.0f);
}

// present a vector of input cases to the network nnet and do forward propagation.
// the dataset is assumed to contain a number of cases equal to
// the nCases in the network
void PresentInputsFromDataSet(MLPNetwork *nnet, DataSet *dset, int actf)
{
    // transfer data to device (if it's not already there)
    if (!TransferDataSetToDevice(dset)) {
        fprintf(stderr, "Could not transfer data set to device\n");
        return;
    }

    // do forward propagation
    PresentInputs(nnet, dset->d_inputs, actf);
}

// present a vector of input cases to the network nnet and do forward propagation.
// d_inputs is assumed to be in device memory, and of size equal to N * nnet->nCases,
// where N is the number of inputs to the network
void PresentInputs(MLPNetwork *nnet, float *d_inputs, int actf)
{
    nnet->layers[0]->d_outs = d_inputs;

    int nn;
    for (int l = 1; l < nnet->nLayers; ++l) {
        nn = nnet->layers[l]->nNeurons;
        if (actf == ACTF_THRESHOLD)
            forward_layer_threshold<<<nnet->nCases, nn>>>(nnet->d_weights,
                                                nnet->layers[l]->weightOffset,
                                                nnet->layers[l]->weightsPerNeuron,
                                                nnet->layers[l-1]->d_outs,
                                                nnet->layers[l-1]->nNeurons,
                                                nnet->layers[l]->d_outs);
        else
            forward_layer<<<nnet->nCases, nn>>>(nnet->d_weights,
                                                nnet->layers[l]->weightOffset,
                                                nnet->layers[l]->weightsPerNeuron,
                                                nnet->layers[l-1]->d_outs,
                                                nnet->layers[l-1]->nNeurons,
                                                nnet->layers[l]->d_outs);
    }
    
}

bool PrepareForTesting(MLPNetwork *nnet, int nCases)
{
    if (nnet->nCases != nCases) {
        nnet->nCases = nCases;
        return ReallocateOutputs(nnet);
    }

    // no need to reallocate outputs
    return true;
}

// ------------------------------------------------------------------------
// --- backpropagation ----------------------------------------------------
// ------------------------------------------------------------------------

// Calculate the deltas for each neuron in the output layer, and the
// error between the actual and expected outputs
//
// grid should be <<<Nc, Nn>>> for Nc cases and Nn neurons in layer
__global__ void deltas_output(float *outs, float *expected, float *d_deltas,
                              float *err)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    err[tid] = expected[tid] - outs[tid];
    d_deltas[tid] = -err[tid] * dsigmoid(outs[tid]);
}

// Calculate the deltas for each neuron in a hidden layer
//
// grid should be <<<Nc, Nn>>> for Nc cases and Nn neurons in layer
__global__ void deltas_hlayer(float *outs, float *d_weights, float *d_deltas,
                              float *d_dltnext, int neuronsNext,
                              int nxtLayerWOffset, int weightsPerNeuronNxt)
{
    // index for delta being calculated on hidden layer
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    // index for first delta on next layer
    int oid = blockIdx.x * neuronsNext;
    // index for relevant weights (neurons in next layer)
    int wid = nxtLayerWOffset + threadIdx.x + 1;  // +1 to account for bias weight

    d_deltas[tid] = 0.0f;
    for (int i = 0; i < neuronsNext; ++i, wid += weightsPerNeuronNxt)
        d_deltas[tid] += d_weights[wid] * d_dltnext[oid+i] * dsigmoid(outs[tid]);
}


// Calculate the derivatives of the error relative to each weight
//
// grid should be <<<Nc, Nw>>> for Nc cases and Nw total weights for layer
__global__ void derivs_layer(float *d_inputs, float *d_deltas, float *d_derivs,
			     int nNeurons, int neuronsPrev, 
			     int nWeights, int weightsPerNeuron, int weightOffset)
{
    // weight index
    int wid = blockIdx.x * nWeights + weightOffset + threadIdx.x;
    // delta index
    int did = blockIdx.x * nNeurons + (threadIdx.x / weightsPerNeuron);
    // input index
    int iid = blockIdx.x * neuronsPrev + (threadIdx.x % weightsPerNeuron) - 1;

    float inp = (threadIdx.x % weightsPerNeuron == 0? 1.0f : d_inputs[iid]);

    d_derivs[wid] = d_deltas[did] * inp;
}

//__global__ sum_derivs(float *d_derivs)
//{
//}

// launch grid: <<<1, NWEIGHTS>>> for number of weights
// TODO: do a proper reduction instead of a set number of sums
__global__ void update_weights_nreduc(float *d_weights, float *d_derivs, float lrate,
                                      int nCases, int nWeights)
{
    float dE = 0.0f;
    int wid = blockIdx.x * blockDim.x + threadIdx.x;

    // sum all derivs for the same weight
    for (int i = 0; i < nCases; ++i)
	dE += d_derivs[i * nWeights + wid];

    // update weight
    d_weights[wid] -= (lrate * dE);
}

bool PrepareForTraining(MLPNetwork *nnet, DataSet *trainData)
{
    if (trainData->nCases != nnet->nCases) {
        nnet->nCases = trainData->nCases;
        if (!ReallocateOutputs(nnet))
            return false;

        if (!ReallocateDeltas(nnet)) {
            FreeOutputs(nnet);
            return false;
        }
    }

    return true;
}

float BatchTrainBackprop(MLPNetwork *nnet, DataSet *data, int epochs,
                         float lrate, int calcSSE, int printSSE)
{
    float *err = NULL, *d_err = NULL;
    float *d_derivs;
    float sse = 0.0f;
    MLPLayer *outLayer = nnet->layers[nnet->nLayers - 1];
    int nOutputs = outLayer->nNeurons;

    if (!PrepareForTraining(nnet, data))
        return -1.0f;
    
    if (!TransferDataSetToDevice(data))
        return -1.0f;
    
    // allocate space for errors
    d_err = allocateFloatsDev(nOutputs * data->nCases);

    if (d_err == NULL) {
        fprintf(stderr, "Error allocating memory on device to store errors: %s\n",
                hipGetErrorString(hipGetLastError()));
        exit(-1);
    }
    
    if (calcSSE) {
        err = (float*) malloc(nOutputs * data->nCases * sizeof(float));
        if (err == NULL) {
            fprintf(stderr, "Couldn't allocate memory to store errors\n.");
            exit(-1);
        }
    }

    // allocate memory for derivatives
    d_derivs = allocateFloatsDev(data->nCases * nnet->nWeights);
    
    for (int e = 0; e < epochs; ++e) {        
        // forward propagation of all the cases
        PresentInputs(nnet, data->d_inputs, ACTF_SIGMOID);
        //hipDeviceSynchronize();
        
        // // print outputs (debug)
        // float *outs = (float*) malloc(data->nCases * nOutputs * sizeof(float));
        // CopyNetworkOutputs(nnet, outs);
        // for (int i = 0; i < data->nCases * nOutputs; ++i)
        //     printf("%5.3f ", outs[i]);
        // printf("|| "); printf("\n");
        // free(outs);
        // // print outputs (debug end)
        
        // backpropagation: calculation of deltas
        deltas_output<<<data->nCases, nOutputs>>>(outLayer->d_outs,
                                                  data->d_outputs,
                                                  outLayer->d_deltas,
                                                  d_err);

        // // print deltas for output layer (debug)
        // float *deltas = (float*) malloc(data->nCases * nOutputs * sizeof(float));
        // hipMemcpy(deltas, outLayer->d_deltas, data->nCases * nOutputs * sizeof(float), hipMemcpyDeviceToHost);
        // for (int i = 0; i < data->nCases * nOutputs; ++i)
        //     printf("%5.3f ", deltas[i]);
        // printf(" -- ");
        // free(deltas);
        // // (debug end)
            

        MLPLayer *layer;
        MLPLayer *nextLayer = outLayer;
        for (int l = nnet->nLayers-2; l > 0; --l) {
            layer = nnet->layers[l];
            deltas_hlayer<<<data->nCases, layer->nNeurons>>>(layer->d_outs,
                                                             nnet->d_weights,
                                                             layer->d_deltas,
                                                             nextLayer->d_deltas,
                                                             nextLayer->nNeurons,
                                                             nextLayer->weightOffset,
                                                             nextLayer->weightsPerNeuron);
            nextLayer = layer;

            // // print deltas for layer (debug)
            // deltas = (float*) malloc(data->nCases * layer->nNeurons * sizeof(float));
            // hipMemcpy(deltas, layer->d_deltas, data->nCases * layer->nNeurons * sizeof(float), hipMemcpyDeviceToHost);
            // for (int i = 0; i < data->nCases * layer->nNeurons; ++i)
            //     printf("%5.3f ", deltas[i]);
            // printf(" -- ");
            // free(deltas);
            // // (debug end)           
        }
        
        // calculate SSE for this epoch
	if (calcSSE) {
	    sse = 0.0f;
	    hipMemcpy(err, d_err, data->nCases * nOutputs * sizeof(float), hipMemcpyDeviceToHost);
	    for (int i = 0; i < data->nCases * nOutputs; ++i) {
		//printf("%6.3f ", err[i]);
		sse += (err[i] * err[i]);
	    }

	    if (printSSE)
		printf("- SSE = %5.3f\n", sse);
	}

        // calculate derivatives of the error
        MLPLayer *prevLayer = nnet->layers[0];
        int nw;
        for (int l = 1; l < nnet->nLayers; ++l) {
            layer = nnet->layers[l];
            nw = layer->nNeurons * layer->weightsPerNeuron;
            derivs_layer<<<data->nCases, nw>>>(prevLayer->d_outs,
                                               layer->d_deltas,
                                               d_derivs,
                                               layer->nNeurons,
                                               prevLayer->nNeurons,
                                               nnet->nWeights,
                                               layer->weightsPerNeuron,
                                               layer->weightOffset);
            prevLayer = layer;
        }

        // update weights based on derivatives
        update_weights_nreduc<<<1, nnet->nWeights>>>(nnet->d_weights, d_derivs,
                                                     lrate, data->nCases,
                                                     nnet->nWeights);
    }
    
    if (err != NULL)
        free(err);
    
    // cleanup
    hipFree(d_err);
    hipFree(d_derivs);
    FreeDeltas(nnet);

    hipDeviceSynchronize();
    return sse;
}

// ------------------------------------------------------------------------
// --- utility functions --------------------------------------------------
// ------------------------------------------------------------------------

// Copy the outputs for network nnet, stored in device memory, to
// host memory pointed to by outs. outs must have size equal to N * nnet->nCases,
// where N is the number of output neurons in the network
bool CopyNetworkOutputs(MLPNetwork *nnet, float *outs)
{
    hipError_t e;
    
    MLPLayer *last = nnet->layers[nnet->nLayers-1];
    
    e = hipMemcpy(outs, last->d_outs,
                   last->nNeurons * nnet->nCases * sizeof(float),
                   hipMemcpyDeviceToHost);

    if (e != hipSuccess) {
        fprintf(stderr, "Error copying outputs from device to host: %s\n",
                hipGetErrorString(e));
        return false;
    }

    return true;
}

void PrintWeights(MLPNetwork *nnet)
{
    float *h_weights;
    hipError_t e;

    h_weights = (float*) malloc(nnet->nWeights * sizeof(float));

    if (h_weights == NULL) {
        printf("Error allocating host memory to copy weights.\n");
    }
    else {
        e = hipMemcpy(h_weights, nnet->d_weights, nnet->nWeights * sizeof(float),
                       hipMemcpyDeviceToHost);

        if (e != hipSuccess) {
            fprintf(stderr, "Error copying weights from device to host: %s\n", 
                    hipGetErrorString(e));
        }
        
        for (int i = 0; i < nnet->nWeights; ++i) {
            printf("%4.5f ", h_weights[i]);
        }
        printf("\n");        
    }

    free(h_weights);
}

// return an array of floats with the outputs for layer with index ixLayer
float *GetLayerOutputs(MLPNetwork *nnet, int ixLayer)
{
    int   length = nnet->layers[ixLayer]->nNeurons * nnet->nCases;
    float *result = (float*) malloc(length * sizeof(float));

    if (result == NULL)
        return NULL;

    // TODO: check hipMemcpy for errors
    hipMemcpy(result, nnet->layers[ixLayer]->d_outs,
               length * sizeof(float), hipMemcpyDeviceToHost);

    return result;
}
