#include "hip/hip_runtime.h"
/*

  iris.cu
  Classification of the iris dataset from Fisher using neural networks
  implemented in CUDA. 

  Andrei de A. Formiga, 2012-05-21

 */

#include <stdio.h>
#include <stdlib.h>
#include <string.h>

#include "mlpnnets.h"

#define SEED                  631814ULL

#define MAX_ABS               1.2f

#define EPOCHS                7000
#define LEARNING_RATE         0.003f

#define MAX(a, b)             (a >= b? a : b)

// neurons per layer (4 inputs, 8 hidden, 3 outputs)
int neuronsPerLayer[] = { 4, 8, 3 };

typedef enum tagClass {
    iris_setosa,
    iris_versicolor,
    iris_virginica
} Class;

DataSet* read_dataset(char *filename)
{
    FILE    *f;
    int     i, j;
    float   slen, swid, plen, pwid;
    char    buffer[140];
    DataSet *dset;

    f = fopen(filename, "r");
    if (f == NULL) {
        fprintf(stderr, "File not found: %s\n", filename);
        return NULL;
    }

    // count lines in file to allocate dataset arrays
    i = 0;
    while (fgets(buffer, 140, f) != NULL)
        ++i;

    if (!feof(f) || ferror(f)) {
        fprintf(stderr, "IO error while reading from file\n");
        fclose(f);
        return NULL;
    }
    fseek(f, 0, SEEK_SET);

    dset = CreateDataSet(i, 4, 3);

    if (dset == NULL) {
        fprintf(stderr, "Error creating dataset\n"); 
        return NULL;
    }

    int  iix = 0, oix = 0;
    bool done = false;
    while (!done) {
        j = fscanf(f, "%f,%f,%f,%f,%s\n", &slen, &swid,
                   &plen, &pwid, buffer);

        if (j != 5)
            done = true;
        else {
            //printf("%f, %f, %f, %f\n", slen, swid, plen, pwid);
            dset->inputs[iix++] = slen;
            dset->inputs[iix++] = swid;
            dset->inputs[iix++] = plen;
            dset->inputs[iix++] = pwid;

            if (strstr(buffer, "setosa")) {
                dset->outputs[oix++] = 0.9f;
                dset->outputs[oix++] = 0.1f;
                dset->outputs[oix++] = 0.1f;
            } else if (strstr(buffer, "versicolor")) {
                dset->outputs[oix++] = 0.1f;
                dset->outputs[oix++] = 0.9f;
                dset->outputs[oix++] = 0.1f;
            } else { // assume class "virginica"
                dset->outputs[oix++] = 0.1f;
                dset->outputs[oix++] = 0.1f;
                dset->outputs[oix++] = 0.9f;
            }
        }
    }

    fclose(f);

    return dset;
}

void print_dataset(DataSet *dset)
{
    int i, j;

    printf("Number of cases: %d\n", dset->nCases);
    for (i = 0; i < dset->nCases; ++i) {
        for (j = 0; j < dset->inputSize; ++j)
            printf("%3.2f ", dset->inputs[i*dset->inputSize+j]);
        printf(" | ");
        for (j = 0; j < dset->outputSize; ++j)
            printf("%3.2f ", dset->outputs[i*dset->outputSize+j]);
        printf("\n");
    }
}

Class output_to_class(float *output)
{
    double max;

    max = MAX(output[0], MAX(output[1], output[2]));
    if (output[0] == max)
        return iris_setosa;
    else if (output[1] == max)
        return iris_versicolor;

    return iris_virginica;
}


char    *setosa = "setosa";
char    *versicolor = "versicolor";
char    *virginica = "virginica";

char *class_to_string(Class c)
{
    char *res;

    switch(c) {
    case iris_setosa:
        res = setosa;
        break;

    case iris_versicolor:
        res = versicolor;
        break;

    default:
        res = virginica;
    }

    return res;
}

void print_network_data(MLPNetwork *net)
{
    printf("nLayers = %d, d_weights = %lu, nWeights = %d, nCases = %d\n",
           net->nLayers, (unsigned long) net->d_weights, net->nWeights, net->nCases);
    printf("output ptr for first layer: %lu\n", (unsigned long) net->layers[0]->d_outs);
    printf("output ptr for last layer: %lu\n", (unsigned long) net->layers[net->nLayers-1]->d_outs);
}

int main(int argc, char **argv)
{
    int     i;
    int     errors;
    DataSet *train_set;
    DataSet *test_set;
    float   e;
    double  acc;
    Class   predicted, desired;

    MLPNetwork *irisnn;

    // training
    train_set = read_dataset("iris.train");

    if (train_set == NULL) {
        fprintf(stderr, "Error reading training set\n");
        exit(1);
    }

    irisnn = CreateNetwork(3, neuronsPerLayer);
    RandomWeights(irisnn, MAX_ABS, SEED);

    printf("Training network with %d epochs...\n", EPOCHS);
    e = BatchTrainBackprop(irisnn, train_set, EPOCHS, LEARNING_RATE, 0, 0);
    printf("Training finished, approximate final SSE: %f\n", e);

    printf("Weights after training:\n");
    PrintWeights(irisnn);

    printf("-----------------------------------------\n");

    // free the training dataset
    hipDeviceSynchronize();
    DestroyDataSet(train_set);

    // testing
    test_set = read_dataset("iris.test");

    if (test_set == NULL) {
        fprintf(stderr, "Error reading test set\n");
        return -1;
    }

    errors = 0;

    if (!PrepareForTesting(irisnn, test_set->nCases)) {
        fprintf(stderr, "Error preparing network for testing\n");
        return -1;
    }

    printf("Testing with %d cases...\n", test_set->nCases);
    PresentInputsFromDataSet(irisnn, test_set, ACTF_SIGMOID);

    hipDeviceSynchronize();

    printf("Weights again:\n");
    PrintWeights(irisnn);    

    float *output = (float*) malloc(sizeof(float) * test_set->nCases * test_set->outputSize);

    if (output == NULL) {
        fprintf(stderr, "Could not allocate memory for copying output to host\n");
        return -1;
    }

    if (!CopyNetworkOutputs(irisnn, output)) {
        fprintf(stderr, "Could not get device outputs\n");
        return -1;
    }

    for (i = 0; i < test_set->nCases; ++i) {
        predicted = output_to_class(output + (i * test_set->outputSize));
        desired = output_to_class(test_set->outputs + (i * test_set->outputSize));
        if (predicted != desired)
            ++errors;
        printf("Case %d | predicted: %s, desired: %s, outputs: %4.3f %4.3f %4.3f\n", i,
               class_to_string(predicted), class_to_string(desired),
               output[i*test_set->outputSize], output[i*test_set->outputSize+1], 
               output[i*test_set->outputSize+2]);
    }

    free(output);

    acc = 100.0 - (100.0 * errors / test_set->nCases);
    printf("Testing accuracy: %f\n", acc);
    printf("Total classificarion errors: %d\n", errors);

    DestroyNetwork(irisnn);
    DestroyDataSet(test_set);

    return 0;
}
