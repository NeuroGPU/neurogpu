
/* 
   xor_pcases_ref.cu
   Implementation of a XOR neural network in CUDA, 
   calculating output of many input cases in parallel.
   (Refactored version.)

   Andrei de A. Formiga, 2012-03-31
*/


#include <hip/hip_runtime.h>
#include <stdio.h>


// weights for the hidden layer
float weights_h[] = { 0.5f, -1.0f, -1.0f,
                      -1.5f, 1.0f, 1.0f };

float weights_h2[2][3] = 
    { 
{ 0.5f, -1.0f, -1.0f }, 
{ -1.5f, 1.0f, 1.0f } };

// weights for the output layer
float weights_o[] = { 0.5f, -1.0f, -1.0f };

// weight arrays for the device
float *dev_hw;
float *dev_ow;

// device input
float *dev_in;

// device hidden outputs
float *dev_hidden;

// device output
float *dev_out;

// inputs
float inputs[] = { 0.0f, 0.0f, 0.0f, 1.0f,
                   1.0f, 0.0f, 1.0f, 1.0f };

const int ncases = 4;
const int input_size = 2;
const int hidden_size = 2;

// weights per neuron
const int ws_per_node[2] = { 3, 2 };

int *dev_wpn;

// index weight from node j to node i in layer l, using array ws of weights
#define W(ws, l, i, j)   ( ws[i * dev_wpn[l] + j] )

// desired outputs
float outputs[] = { 0.0f, 1.0f, 1.0f, 0.0f };

// kernel for hidden layer (indexed as layer 0)
__global__ void calculate_hidden(float *dev_hw, int *dev_wpn, float *input, float *hidden)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int input_ix = blockIdx.x * blockDim.x;
    int node = threadIdx.x;
    float h;

    // h = dev_hw[toff * 3] * 1.0f +
    //     dev_hw[toff * 3 + 1] * input[input_ix] +
    //     dev_hw[toff * 3 + 2] * input[input_ix+1];

    h = W(dev_hw, 0, node, 0) * 1.0f +
        W(dev_hw, 0, node, 1) * input[input_ix] + 
        W(dev_hw, 0, node, 2) * input[input_ix+1];

    // threshold
    if (h > 0.0f)
        hidden[tid] = 1.0f;
    else
        hidden[tid] = 0.0;
}

// kernel for output layer
__global__ void calculate_output(float *dev_ow, float *hidden, float *output)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int hidden_ix = blockIdx.x * blockDim.x;
    int toff = threadIdx.x;    
    float o;

    o = dev_ow[toff] * 1.0f +
        dev_ow[toff+1] * hidden[2*hidden_ix] +
        dev_ow[toff+2] * hidden[2*hidden_ix+1];

    // threshold
    if (o > 0.0f)
        output[tid] = 1.0f;
    else
        output[tid] = 0.0f;
}

int main(int argc, char **argv)
{
    float out[ncases];

    printf("### XOR test (forward propagation)\n");
    
    hipMalloc((void**) &dev_hw, 6 * sizeof(float));
    hipMalloc((void**) &dev_ow, 3 * sizeof(float));
    hipMalloc((void**) &dev_in, ncases * input_size * sizeof(float));
    hipMalloc((void**) &dev_hidden, ncases * hidden_size * sizeof(float));
    hipMalloc((void**) &dev_out, ncases * sizeof(float));  // output size = 1
    hipMalloc((void**) &dev_wpn, 2 * sizeof(int));
    
    hipMemcpy(dev_hw, weights_h, 6 * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(dev_ow, weights_o, 3 * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(dev_wpn, ws_per_node, 2 * sizeof(int), hipMemcpyHostToDevice);

    // try inputs
    hipMemcpy(dev_in, inputs, ncases * input_size * sizeof(float), hipMemcpyHostToDevice);
    calculate_hidden<<<4, 2>>>(dev_hw, dev_wpn, dev_in, dev_hidden);
    calculate_output<<<4, 1>>>(dev_ow, dev_hidden, dev_out);
    hipMemcpy(out, dev_out, ncases * sizeof(float), hipMemcpyDeviceToHost);

    for (int i = 0; i < ncases; ++i)
        printf("Input: %2.1f %2.1f -- Output: %f\n", inputs[input_size*i],
               inputs[input_size*i+1], out[i]);
    
    hipFree(dev_hw);
    hipFree(dev_ow);
    hipFree(dev_in);
    hipFree(dev_hidden);
    hipFree(dev_out);
    
    return 0;
}
